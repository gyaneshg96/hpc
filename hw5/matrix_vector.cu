
#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>


#define BLOCK_SIZE 1024

void matrix_product(double* dot_ptr, const double* x, const double* y, long m, long n){
  // #pragma omp parallel for schedule(static) reduction(+:sum)
  for (long i = 0; i < m; i++){
    double sum = 0;
    for(long j = 0; j < n; j++)
      sum += x[i*n+j] *y[j];
  dot_ptr[i] = sum;
  }
}

void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

__global__ void matrix_product_kernel(double *dotptr, const double*x, double *y, long N, int flag){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N && flag == 1) smem[threadIdx.x] = x[idx]*y[idx];
  else if (idx < N && flag == 0) smem[threadIdx.x] = x[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) dotptr[blockIdx.x] = smem[0] + smem[1];
  }
}

double error(double *a, double *b, long n){
  double err = 0.0; 
  for(long i = 0; i < n;i++){
    err += fabs(a[i] - b[i]);
  }
  return err;
}

int main() {
  
  long m = 1000;
  long n = 100000;

  double *x, *y, *prod_ref;
  hipHostMalloc((void**)&x, m*n * sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&y, n * sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&prod_ref, m * sizeof(double), hipHostMallocDefault);

  #pragma omp parallel for schedule(static)
  for (long i = 0; i < m*n; i++) {
    x[i] = 1.0/(i+1);
  }

  for(long i = 0; i <n;i++){
    y[i] = 1.0/(n-i);
  }

  double tt = omp_get_wtime();

  matrix_product(prod_ref, x,y, m,n);

  printf("CPU Bandwidth = %f GB/s\n", (m+1)*(n+1)*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  double *x_d, *y_d, *prod, *z_d, *prod_d;
  hipMalloc(&x_d, m*n*sizeof(double));
  hipMalloc(&y_d, n*sizeof(double));
  hipHostMalloc((void**)&prod, m * sizeof(double), hipHostMallocDefault);


  long N_work = 1;
  for (long i = (n+BLOCK_SIZE-1)/(BLOCK_SIZE); i > 1; i = (i+BLOCK_SIZE-1)/(BLOCK_SIZE)) N_work += i;
  hipMalloc(&z_d, N_work*sizeof(double)); // extra memory buffer for reduction across thread-blocks
  hipMalloc(&prod_d, m*sizeof(double)); // extra memory buffer for reduction across thread-blocks

  hipMemcpyAsync(x_d, x, m*n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(y_d, y, n*sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  tt = omp_get_wtime();

  for (long i = 0; i < m;i++){
    double* tempdot = z_d;
    
    long Nb = (n+BLOCK_SIZE-1)/(BLOCK_SIZE);
    matrix_product_kernel<<<Nb,BLOCK_SIZE>>>(tempdot, x_d + i*n, y_d, n, 1);
    while (Nb > 1) {
    long N = Nb;
    Nb = (Nb+BLOCK_SIZE-1)/(BLOCK_SIZE);
    matrix_product_kernel<<<Nb,BLOCK_SIZE>>>(tempdot + N, tempdot, y_d, N, 0);
    tempdot += N;
    }
    
    //prod_d[i] = *tempdot;
    hipMemcpy(prod_d + i, tempdot, sizeof(double), hipMemcpyDeviceToDevice);
  }

  hipMemcpyAsync(prod, prod_d, m*sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  printf("GPU Bandwidth = %f GB/s\n", (m+1)*(n+1)*sizeof(double) / (omp_get_wtime()-tt)/1e9);
  printf("Error = %f\n", error(prod, prod_ref, m));

  hipFree(x_d);
  hipFree(y_d);
  hipFree(z_d);
  hipHostFree(x);
  hipHostFree(y);
}
