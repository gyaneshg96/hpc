
#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>

void dot_product(double* dot_ptr, const double* x, const double* y, long N){
  double sum = 0;
  #pragma omp parallel for schedule(static) reduction(+:sum)
  for (long i = 0; i < N; i++) sum += x[i]*y[i];
  *dot_ptr = sum;
}

void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

#define BLOCK_SIZE 1024

__global__ void dotproduct_kernel2(double* sum, const double* a, const double*b, long N, int flag){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N && flag == 1) smem[threadIdx.x] = a[idx]*b[idx];
  else if (idx < N && flag == 0) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}

int main() {
  long N = (1UL<<25);

  double *x, *y;
  hipHostMalloc((void**)&x, N * sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&y, N * sizeof(double), hipHostMallocDefault);
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N; i++) {
    x[i] = 1.0/(i+1);
    y[i] = 1.0/(N - i);
  }

  double dot_ref, dot;
  double tt = omp_get_wtime();
  dot_product(&dot_ref, x,y, N);
  printf("CPU Bandwidth = %f GB/s\n", 1*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  double *x_d, *y_d, *z_d;
  hipMalloc(&x_d, N*sizeof(double));
  hipMalloc(&y_d, N*sizeof(double));
  long N_work = 1;
  for (long i = (N+BLOCK_SIZE-1)/(BLOCK_SIZE); i > 1; i = (i+BLOCK_SIZE-1)/(BLOCK_SIZE)) N_work += i;
  hipMalloc(&z_d, N_work*sizeof(double)); // extra memory buffer for reduction across thread-blocks

  hipMemcpyAsync(x_d, x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(y_d, y, N*sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  tt = omp_get_wtime();


  double* dot_d = z_d;
  long Nb = (N+BLOCK_SIZE-1)/(BLOCK_SIZE);
  dotproduct_kernel2<<<Nb,BLOCK_SIZE>>>(dot_d, x_d, y_d, N, 1);
  while (Nb > 1) {
    long N = Nb;
    Nb = (Nb+BLOCK_SIZE-1)/(BLOCK_SIZE);
    dotproduct_kernel2<<<Nb,BLOCK_SIZE>>>(dot_d + N, dot_d, y_d, N, 0);
    dot_d += N;
  }



  hipMemcpyAsync(&dot, dot_d, 1*sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  printf("GPU Bandwidth = %f GB/s\n", 1*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);
  printf("Error = %f\n", fabs(dot-dot_ref));

  hipFree(x_d);
  hipFree(y_d);
  hipFree(z_d);
  hipHostFree(x);
  hipHostFree(y);

  return 0;
}
